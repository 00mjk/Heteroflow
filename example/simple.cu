#include "hip/hip_runtime.h"
#include <iostream>
#include <heteroflow/heteroflow.hpp>

__global__ void simple(float* X, size_t Nx, float* Y, size_t Ny) {
}

int main() {

  hf::TaskBase t1, t2, t3, t4;

  t1.precede(&t2, &t2, &t3);

  std::cout << t1.num_dependents() << " " << t1.num_successors() << std::endl;

  float* h_X = nullptr; 
  float* h_Y;

  //HF_THROW_IF(h_X == nullptr, "f---", h_X)
  HF_CHECK_CUDA(hipErrorNotInitialized, "succeFFFFFF", h_X);
  //success                            = hipSuccess,
  //missing_configuration              = hipErrorMissingConfiguration,
  //memory_allocation                  = hipErrorOutOfMemory,
  //initialization_error               = hipErrorNotInitialized,
  
  // create a heteroflow
  /*hf::Heteroflow hf;

  auto new_X = hf.host([&](){ h_X = new float [32]; });
  auto new_Y = hf.host([&](){ h_Y = new float [64]; });
  auto gpu_X = hf.pull(h_X, n_X);
  auto gpu_Y = hf.pull(h_Y, n_Y);

  // kernel task (depends on gpu_X and gpu_Y)
  auto kernel = hf.kernel(simple, gpu_X, 32, gpu_Y, 64);

  auto push_X = hf.push(h_X, gpu_X, n_X);
  auto push_Y = hf.push(h_Y, gpu_Y, n_Y);
  auto kill_X = hf.host([&](){ delete [] h_X; });
  auto kill_Y = hf.host([&](){ delete [] h_Y; });
  
  // build up the dependency
  new_X.precede(gpu_X);
  new_Y.precede(gpu_Y);
  gpu_X.precede(kernel);
  gpu_Y.precede(kernel);
  kernel.precede(push_X, push_Y);
  push_X.precede(kill_X);
  push_Y.precede(kill_Y);
  
  // dump the heteroflow graph
  hf.dump(std::cout);
  
  // create an executor
  hf::Executor executor;
  executor.run(hf);
*/
  return 0;
}





