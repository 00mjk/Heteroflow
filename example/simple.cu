#include "hip/hip_runtime.h"
#include <iostream>
#include <heteroflow/heteroflow.hpp>

__global__ void simple(float* X, size_t Nx, float* Y, size_t Ny) {
  printf(
    "Hello from block %d, thread %d (%lux%lu)\n", 
    blockIdx.x, threadIdx.x,
    Nx, Ny
  );
}

__global__ void hello_kernel(int id) {
  printf(
    "Hello from block %d, thread %d (my-id=%d)\n", 
    blockIdx.x, threadIdx.x, id
  );
}
  
int main() {

  // create a heteroflow
  hf::Heteroflow hf("simple");

  float* h_X {nullptr};
  float* h_Y {nullptr};
  size_t n_X {100};
  size_t n_Y {200};

  auto new_X = hf.host([&](){ h_X = new float [n_X]; }).name("host_X");
  auto new_Y = hf.host([&](){ h_Y = new float [n_Y]; }).name("host_Y");
  auto gpu_X = hf.pull(h_X, n_X*sizeof(float)).name("pull_X");
  auto gpu_Y = hf.pull(h_Y, n_Y*sizeof(float)).name("pull_Y");

  // kernel task (depends on gpu_X and gpu_Y)
  auto kernel = hf.kernel(simple, gpu_X, n_X, gpu_Y, n_Y).name("kernel");

  auto push_X = hf.push(h_X, gpu_X, n_X*sizeof(float)).name("push_X");
  auto push_Y = hf.push(h_Y, gpu_Y, n_Y*sizeof(float)).name("push_Y");
  auto kill_X = hf.host([&](){ delete [] h_X; }).name("kill_X");
  auto kill_Y = hf.host([&](){ delete [] h_Y; }).name("kill_Y");

  // build up the dependency
  new_X.precede(gpu_X);
  new_Y.precede(gpu_Y);
  gpu_X.precede(kernel);
  gpu_Y.precede(kernel);
  kernel.precede(push_X, push_Y);
  push_X.precede(kill_X);
  push_Y.precede(kill_Y);
  
  // dump the heteroflow graph
  hf.dump(std::cout); 

  //auto A = hf.host([](){std::cout << "A\n";});
  //auto B = hf.host([](){std::cout << "B\n";});
  //auto C = hf.host([](){std::cout << "C\n";});
  //auto D = hf.host([](){std::cout << "D\n";});
  //
  //A.precede(B);
  //A.precede(C);
  //B.precede(D);
  //C.precede(D);

  ////auto k1 = hf.kernel(hello_kernel, 1).name("kernel1");
  //int* ptr1 = new int [100];
  //int* ptr2 = new int [100];
  //for(int i=0; i<100; ++i) {
  //  ptr1[i] = 9;
  //  ptr2[i] = 0;
  //}
  //auto p1 = hf.pull(ptr1, 100*sizeof(int)).name("pull");
  //auto p2 = hf.push(ptr2, p1, 100*sizeof(int));
  //p1.precede(p2);
  
  // create an executor
  hf::Executor executor(1, 1);
  
  executor.run(hf).wait();

  //for(int i=0; i<100; ++i) {
  //  assert(ptr2[i] == ptr1[i]);
  //}

  //hipDeviceSynchronize();

  return 0;
}





